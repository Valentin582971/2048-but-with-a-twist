#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#define GRID_SIZE 4

// Weights for monte carlo simulations and the steps :
#define EMPTY_CELLS_WEIGHT 50000
#define TOTAL_SUM_WEIGHT 1
#define SMOOTHNESS_WEIGHT 100
#define MAX_TILE_WEIGHT 20

#define STEPS 5  // Number of Monte Carlo simulations

// Uncomment the following macros to enable specific features
#define USE_AI    // Enable AI to play the game
// #define AZERTY    // Set player controls to AZERTY (ZQSD)

using namespace std;

// CUDA kernel to move left
__global__ void move_left(int* grid, int size) {
    int row = blockIdx.x;
    for (int col = 0; col < size; ++col) {
        if (grid[row * size + col] == 0) {
            for (int k = col + 1; k < size; ++k) {
                if (grid[row * size + k] != 0) {
                    grid[row * size + col] = grid[row * size + k];
                    grid[row * size + k] = 0;
                    break;
                }
            }
        }
    }
    for (int col = 0; col < size - 1; ++col) {
        if (grid[row * size + col] == grid[row * size + col + 1] && grid[row * size + col] != 0) {
            grid[row * size + col] *= 2;
            grid[row * size + col + 1] = 0;
        }
    }
    for (int col = 0; col < size; ++col) {
        if (grid[row * size + col] == 0) {
            for (int k = col + 1; k < size; ++k) {
                if (grid[row * size + k] != 0) {
                    grid[row * size + col] = grid[row * size + k];
                    grid[row * size + k] = 0;
                    break;
                }
            }
        }
    }
}

// CUDA kernel to move right
__global__ void move_right(int* grid, int size) {
    int row = blockIdx.x;
    for (int col = size - 1; col >= 0; --col) {
        if (grid[row * size + col] == 0) {
            for (int k = col - 1; k >= 0; --k) {
                if (grid[row * size + k] != 0) {
                    grid[row * size + col] = grid[row * size + k];
                    grid[row * size + k] = 0;
                    break;
                }
            }
        }
    }
    for (int col = size - 1; col > 0; --col) {
        if (grid[row * size + col] == grid[row * size + col - 1] && grid[row * size + col] != 0) {
            grid[row * size + col] *= 2;
            grid[row * size + col - 1] = 0;
        }
    }
    for (int col = size - 1; col >= 0; --col) {
        if (grid[row * size + col] == 0) {
            for (int k = col - 1; k >= 0; --k) {
                if (grid[row * size + k] != 0) {
                    grid[row * size + col] = grid[row * size + k];
                    grid[row * size + k] = 0;
                    break;
                }
            }
        }
    }
}

// CUDA kernel to transpose the grid
__global__ void transpose(int* grid, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int row = idx / size;
    int col = idx % size;

    if (row < col) {
        int tmp = grid[row * size + col];
        grid[row * size + col] = grid[col * size + row];
        grid[col * size + row] = tmp;
    }
}

// CPU function to add a new tile
void add_new_tile(vector<int>& grid) {
    vector<int> empty_positions;
    for (int i = 0; i < grid.size(); ++i) {
        if (grid[i] == 0) empty_positions.push_back(i);
    }
    if (!empty_positions.empty()) {
        int pos = empty_positions[rand() % empty_positions.size()];
        grid[pos] = (rand() % 10 == 0) ? 4 : 2;
    }
}

// Calculate the score for a grid
__host__ int total_score(const vector<int>& grid) {
    int total_sum = 0;
    for (int value : grid) {
        total_sum += value;
    }
    return total_sum;
}

// Evaluate the score for a grid
__host__ int evaluate_grid(const vector<int>& grid) {
    int empty_cells = 0, total_sum = 0, smoothness = 0;
    int max_tile = 0;

    // Check for empty cells and calculate total sum
    for (int i = 0; i < grid.size(); ++i) {
        int value = grid[i];
        if (value == 0) {
            ++empty_cells;
        }
        else {
            total_sum += value;
            max_tile = max(max_tile, value);

            // Calculate smoothness (penalize big differences between neighbors)
            if (i % GRID_SIZE != GRID_SIZE - 1) { // Right neighbor
                int neighbor = grid[i + 1];
                if (neighbor != 0) {
                    smoothness -= abs(value - neighbor);
                }
            }
            if (i < GRID_SIZE * (GRID_SIZE - 1)) { // Downward neighbor
                int neighbor = grid[i + GRID_SIZE];
                if (neighbor != 0) {
                    smoothness -= abs(value - neighbor);
                }
            }
        }
    }

    // Weighted score
    return (empty_cells * EMPTY_CELLS_WEIGHT) +
        (total_sum * TOTAL_SUM_WEIGHT) +
        (smoothness * SMOOTHNESS_WEIGHT) +
        (max_tile * MAX_TILE_WEIGHT);
}


// Monte Carlo function to choose the best move
int monte_carlo_move(const vector<int>& grid, int* d_grid) {
    int best_move = -1;
    double best_score = -1;

    for (int move = 0; move < 4; ++move) {
        double total_score = 0;
        bool valid_move = false;

        for (int step = 0; step < STEPS; ++step) {
            vector<int> temp_grid(grid);
            hipMemcpy(d_grid, temp_grid.data(), GRID_SIZE * GRID_SIZE * sizeof(int), hipMemcpyHostToDevice);

            // Apply the move
            switch (move) {
            case 0: move_left << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE); break;
            case 1: move_right << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE); break;
            case 2:
                transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
                move_left << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE);
                transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
                break;
            case 3:
                transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
                move_right << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE);
                transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
                break;
            }

            // Copy back and check validity
            hipMemcpy(temp_grid.data(), d_grid, GRID_SIZE * GRID_SIZE * sizeof(int), hipMemcpyDeviceToHost);
            if (temp_grid != grid) {
                valid_move = true;
            }
            else {
                break; // No need to continue for an invalid move
            }

            // Add a random tile and evaluate the grid
            add_new_tile(temp_grid);
            total_score += evaluate_grid(temp_grid);
        }

        // Skip invalid moves
        if (!valid_move) continue;

        // Calculate average score for this move
        double average_score = total_score / STEPS;
        if (average_score > best_score) {
            best_score = average_score;
            best_move = move;
        }
    }

    return best_move;
}


// Check if the game is over
bool is_game_over(const vector<int>& grid) {
    // Check for empty cells
    for (int value : grid) {
        if (value == 0) {
            return false;
        }
    }

    // Check for possible horizontal merges
    for (int i = 0; i < GRID_SIZE; ++i) {
        for (int j = 0; j < GRID_SIZE - 1; ++j) {
            int idx = i * GRID_SIZE + j;
            if (grid[idx] == grid[idx + 1]) {
                return false;
            }
        }
    }

    // Check for possible vertical merges
    for (int j = 0; j < GRID_SIZE; ++j) {
        for (int i = 0; i < GRID_SIZE - 1; ++i) {
            int idx = i * GRID_SIZE + j;
            if (grid[idx] == grid[idx + GRID_SIZE]) {
                return false;
            }
        }
    }

    // No empty cells and no possible merges
    return true;
}

// Main function
int main() {
    srand(time(0));

    vector<int> grid(GRID_SIZE * GRID_SIZE, 0);
    int* d_grid;
    hipMalloc(&d_grid, GRID_SIZE * GRID_SIZE * sizeof(int));

    add_new_tile(grid);
    add_new_tile(grid);

    while (!is_game_over(grid)) {
        cout << "Current grid:\n";
        for (int i = 0; i < GRID_SIZE; ++i) {
            for (int j = 0; j < GRID_SIZE; ++j) {
                cout << grid[i * GRID_SIZE + j] << "\t";
            }
            cout << endl;
        }

#ifdef USE_AI
        // AI determines the best move
        int move = monte_carlo_move(grid, d_grid);
        if (move == -1) {
            cout << "No valid moves. Game over." << endl;
            break;
        }

        switch (move) {
        case 0: move_left << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE); break;
        case 1: move_right << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE); break;
        case 2:
            transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
            move_left << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE);
            transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
            break;
        case 3:
            transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
            move_right << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE);
            transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
            break;
        }
#else
        // Player input for manual moves
        char input;
#ifdef AZERTY
        cout << "Enter move (Z=Up, Q=Left, S=Down, D=Right): ";
#else
        cout << "Enter move (W=Up, A=Left, S=Down, D=Right): ";
#endif
        cin >> input;

        bool valid_move = false;
        switch (input) {
#ifdef AZERTY
        case 'Q': case 'q': move_left << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE); valid_move = true; break;
        case 'D': case 'd': move_right << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE); valid_move = true; break;
        case 'Z': case 'z':
            transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
            move_left << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE);
            transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
            valid_move = true;
            break;
        case 'S': case 's':
            transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
            move_right << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE);
            transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
            valid_move = true;
            break;
#else
        case 'A': case 'a': move_left << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE); valid_move = true; break;
        case 'D': case 'd': move_right << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE); valid_move = true; break;
        case 'W': case 'w':
            transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
            move_left << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE);
            transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
            valid_move = true;
            break;
        case 'S': case 's':
            transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
            move_right << <GRID_SIZE, 1 >> > (d_grid, GRID_SIZE);
            transpose << <1, GRID_SIZE* GRID_SIZE >> > (d_grid, GRID_SIZE);
            valid_move = true;
            break;
#endif
        default:
            cout << "Invalid input. Try again.\n";
            break;
        }

        if (!valid_move) continue;
#endif

        hipMemcpy(grid.data(), d_grid, GRID_SIZE * GRID_SIZE * sizeof(int), hipMemcpyDeviceToHost);
        add_new_tile(grid);
    }

    cout << "Game over!" << endl;

    // Print the grid one last time
    for (int i = 0; i < GRID_SIZE; ++i) {
        for (int j = 0; j < GRID_SIZE; ++j) {
            cout << grid[i * GRID_SIZE + j] << "\t";
        }
        cout << endl;
    }
    fprintf(stdout, "Score : ");
    cout << total_score(grid) << endl;
    hipFree(d_grid);


    return 0;
}
